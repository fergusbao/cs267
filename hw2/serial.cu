#include "hip/hip_runtime.h"
#include "common.cuh"
#include <cassert>
#include <iostream>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <vector>

#include <utility>
#include <memory>

// RECOLIC: In cuda, we should take special care to compile extern functions.
//  To make things easy, I'm willing to place all of them in one source.
//  THIS IS NOT AN ERROR!!
#include "common.cu"
// NOTE END

//
//  benchmarking program
//
int main(int argc, char **argv) {
  int navg, nabsavg = 0;
  double davg, dmin, absmin = 1.0, absavg = 0.0;

  if (find_option(argc, argv, "-h") >= 0) {
    printf("Options:\n");
    printf("-h to see this help\n");
    printf("-n <int> to set the number of particles\n");
    printf("-o <filename> to specify the output file name\n");
    printf("-s <filename> to specify a summary file name\n");
    printf("-no turns off all correctness checks and particle output\n");
    return 0;
  }

  int n = read_int(argc, argv, "-n", 1000);

  char *savename = read_string(argc, argv, "-o", NULL);
  char *sumname = read_string(argc, argv, "-s", NULL);

  FILE *fsave = savename ? fopen(savename, "w") : NULL;
  FILE *fsum = sumname ? fopen(sumname, "a") : NULL;

  particle_t *_cuda_managed_particles = nullptr;
  rlib::cuda_assert((hipError_t)hipMallocManaged(&_cuda_managed_particles, n * sizeof(particle_t)));
  particle_t *particles = new(_cuda_managed_particles) particle_t[n]();

  set_size(n);
  init_particles(n, particles);
  double density = 0.0005;
  double cutoff = 0.01;
  double size = sqrt(density * n);
  int sx = floor(size / cutoff + 2);
  int sy = sx;
  std::printf("RDEBUG> grid_size = %d\n", sx);
  //std::vector<int> dict[sx][sy];
  using dict_element_type = std::vector<int>;
  // RECOLIC: FUCKING BRIDGE IS USING GCC 4.8.5 which doesn't support c++14
  //auto _dict_buf_ptr = std::make_unique<dict_element_type[]>(sx*sy);
  auto _dict_buf_ptr = std::unique_ptr<dict_element_type[]>(new dict_element_type[sx*sy]());

#define RLIB_MACRO_ACCESS_2D_DICT(_x, _y) (_dict_buf_ptr[(_x)*sx+(_y)])

  //
  //  simulate a number of time steps
  //
  double simulation_time = read_timer();

  for (int step = 0; step < NSTEPS; step++) {
    navg = 0;
    davg = 0.0;
    dmin = 1.0;
    //
    //  Update bins
    //
    for (int i = 0; i < sx; i++) {
      for (int j = 0; j < sy; j++) {
        RLIB_MACRO_ACCESS_2D_DICT(i, j).clear();
      }
    }

    for (int i = 0; i < n; i++) {
      int a = floor(particles[i].x / cutoff);
      int b = floor(particles[i].y / cutoff);
      RLIB_MACRO_ACCESS_2D_DICT(a, b).push_back(i);
    }
    //
    //  compute forces
    //
    for (int i = 0; i < n; i++) {
      auto &particle = particles[i];

      int a = floor(particle.x / cutoff);
      int b = floor(particle.y / cutoff);

      particle.ax = particle.ay = 0;

      for (int j = 0; j < RLIB_MACRO_ACCESS_2D_DICT(a, b).size(); j++) {
        apply_force(particle, particles[RLIB_MACRO_ACCESS_2D_DICT(a, b)[j]], &dmin, &davg,
                    &navg);
      }
      if (b > 0) {
        for (int j = 0; j < RLIB_MACRO_ACCESS_2D_DICT(a, b - 1).size(); j++) {
          apply_force(particle, particles[RLIB_MACRO_ACCESS_2D_DICT(a, b - 1)[j]], &dmin, &davg,
                      &navg);
        }
      }
      if (b < sy - 1) {
        for (int j = 0; j < RLIB_MACRO_ACCESS_2D_DICT(a, b + 1).size(); j++) {
          apply_force(particle, particles[RLIB_MACRO_ACCESS_2D_DICT(a, b + 1)[j]], &dmin, &davg,
                      &navg);
        }
      }
      if (a > 0) {
        for (int j = 0; j < RLIB_MACRO_ACCESS_2D_DICT(a - 1, b).size(); j++) {
          apply_force(particle, particles[RLIB_MACRO_ACCESS_2D_DICT(a - 1, b)[j]], &dmin, &davg,
                      &navg);
        }
        if (b > 0) {
          for (int j = 0; j < RLIB_MACRO_ACCESS_2D_DICT(a - 1, b - 1).size(); j++) {
            apply_force(particle, particles[RLIB_MACRO_ACCESS_2D_DICT(a - 1, b - 1)[j]], &dmin,
                        &davg, &navg);
          }
        }
        if (b < sy - 1) {
          for (int j = 0; j < RLIB_MACRO_ACCESS_2D_DICT(a - 1, b + 1).size(); j++) {
            apply_force(particle, particles[RLIB_MACRO_ACCESS_2D_DICT(a - 1, b + 1)[j]], &dmin,
                        &davg, &navg);
          }
        }
      }
      if (a < sx - 1) {
        for (int j = 0; j < RLIB_MACRO_ACCESS_2D_DICT(a + 1, b).size(); j++) {
          apply_force(particle, particles[RLIB_MACRO_ACCESS_2D_DICT(a + 1, b)[j]], &dmin, &davg,
                      &navg);
        }
        if (b > 0) {
          for (int j = 0; j < RLIB_MACRO_ACCESS_2D_DICT(a + 1, b - 1).size(); j++) {
            apply_force(particle, particles[RLIB_MACRO_ACCESS_2D_DICT(a + 1, b - 1)[j]], &dmin,
                        &davg, &navg);
          }
        }
        if (b < sy - 1) {
          for (int j = 0; j < RLIB_MACRO_ACCESS_2D_DICT(a + 1, b + 1).size(); j++) {
            apply_force(particle, particles[RLIB_MACRO_ACCESS_2D_DICT(a + 1, b + 1)[j]], &dmin,
                        &davg, &navg);
          }
        }
      }
    }

    //
    //  move particles
    //
    const auto buffer_size = n;
    const auto threads = std::min(n, CUDA_MAX_THREAD_PER_BLOCK);
    const auto blocks = buffer_size / CUDA_MAX_THREAD_PER_BLOCK + 1;
    r267::move_helper<<<blocks, threads>>>(particles, size, buffer_size);
    //for (int i = 0; i < n; i++)
    //  ::move(particles[i]);

    if (find_option(argc, argv, "-no") == -1) {
      //
      // Computing statistical data
      //
      if (navg) {
        absavg += davg / navg;
        nabsavg++;
      }
      if (dmin < absmin)
        absmin = dmin;

      //
      //  save if necessary
      //
      if (fsave && (step % SAVEFREQ) == 0)
        save(fsave, n, particles);
    }
  }
  simulation_time = read_timer() - simulation_time;

  printf("n = %d, simulation time = %g seconds", n, simulation_time);

  if (find_option(argc, argv, "-no") == -1) {
    if (nabsavg)
      absavg /= nabsavg;
    //
    //  -The minimum distance absmin between 2 particles during the run of the
    //  simulation -A Correct simulation will have particles stay at greater
    //  than 0.4 (of cutoff) with typical values between .7-.8 -A simulation
    //  where particles don't interact correctly will be less than 0.4 (of
    //  cutoff) with typical values between .01-.05
    //
    //  -The average distance absavg is ~.95 when most particles are interacting
    //  correctly and ~.66 when no particles are interacting
    //
    printf(", absmin = %lf, absavg = %lf", absmin, absavg);
    if (absmin < 0.4)
      printf("\nThe minimum distance is below 0.4 meaning that some particle "
             "is not interacting");
    if (absavg < 0.8)
      printf("\nThe average distance is below 0.8 meaning that most particles "
             "are not interacting");
  }
  printf("\n");

  //
  // Printing summary data
  //
  if (fsum)
    fprintf(fsum, "%d %g\n", n, simulation_time);

  //
  // Clearing space
  //
  if (fsum)
    fclose(fsum);
  rlib::cuda_assert(hipFree(_cuda_managed_particles));
  if (fsave)
    fclose(fsave);

  return 0;
}
